/*
sjoshi26 shashank joshi
akwatra archit kwatra
vkarri vivek reddy karri
*/


#include <stdlib.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <time.h>


// Variable and constant already defined on the lake_mpi.cu code so defined as extern.

extern int tpdt(double *, double, double);
extern int npebs;
extern int npoints_y;
extern int npoints_x;
extern double end_time;
extern int nthreads;
extern int narea;
extern int numproc, rank;

#define __DEBUG

#ifndef TSCALE
#define TSCALE 1.0
#endif

#ifndef VSQR
#define VSQR 1.0
#endif

/* -----------------Error Check and time Recording setup for GPU side of execution --------------------------------*/

#define CUDA_CALL( err )     __cudaSafeCall( err, __FILE__, __LINE__ )
#define CUDA_CHK_ERR() __cudaCheckError(__FILE__,__LINE__)

/**************************************
* void __cudaSafeCall(cudaError err, const char *file, const int line)
* void __cudaCheckError(const char *file, const int line)
*
* These routines were taken from the GPU Computing SDK
* (http://developer.nvidia.com/gpu-computing-sdk) include file "cutil.h"
**************************************/

inline void __cudaSafeCall( hipError_t err, const char *file, const int line )
{
#ifdef __DEBUG

#pragma warning( push )
#pragma warning( disable: 4127 ) // Prevent warning on do-while(0);
  do
  {
    if ( hipSuccess != err )
    {
      fprintf( stderr, "cudaSafeCall() failed at %s:%i : %s\n",
              file, line, hipGetErrorString( err ) );
      exit( -1 );
    }
  } while ( 0 );
#pragma warning( pop )
#endif  // __DEBUG
  return;
}

inline void __cudaCheckError( const char *file, const int line )
{
#ifdef __DEBUG
#pragma warning( push )
#pragma warning( disable: 4127 ) // Prevent warning on do-while(0);
  do
  {
    hipError_t err = hipGetLastError();
    if ( hipSuccess != err )
    {
      fprintf( stderr, "cudaCheckError() failed at %s:%i : %s.\n",
               file, line, hipGetErrorString( err ) );
      exit( -1 );
    }
    // More careful checking. However, this will affect performance.
    // Comment if not needed.
    /*err = cudaThreadSynchronize();
    if( cudaSuccess != err )
    {
      fprintf( stderr, "cudaCheckError() with sync failed at %s:%i : %s.\n",
               file, line, cudaGetErrorString( err ) );
      exit( -1 );
    }*/
  } while ( 0 );
#pragma warning( pop )
#endif // __DEBUG
  return;
}
/* -----------------Error Check and time Recording setup for GPU side of execution --------------------------------*/




// Device-Specific code to compute f_pebble function.
__device__ double f_pebble(double p, double t)
{
  return -__expf(-TSCALE * t) * p;
}


// GPU - Specific and Rectangular generalization of the evolve13pt function defined in V2, adopted for MPI-CUDA Hybrid.
// 1D Grid and 2D Block Style is used. Threads are alse defined as 2D.
__global__ void evolve13pt_gpu(double *un, double *uc, double *uo, double *pebbles, int n_x, int n_y, double h, double dt, double t){

  int idx_p_1;
  int Neigh;  // North, East, North, South
  int immNeigh; // NorthEast, NorthWest, SouthEast, SouthWest
  int NeighNeigh; // NorthNorth, WestWest, EastEast, SouthSouth

  int i_1;
  int j_1;

  int idx;

  idx_p_1 = blockIdx.x*blockDim.x*blockDim.y + threadIdx.y*blockDim.x+threadIdx.x;

   i_1=idx_p_1/n_y;
   j_1=idx_p_1%n_y;

   idx = (j_1+2) + (i_1+2)*(n_y+4);

   if (idx >= (2*n_y + 2) && idx <= (((n_x+1)*n_y) + n_y + 1)){

     Neigh = uc[idx-1] + uc[idx+1] + uc[idx + n_y + 4] + uc[idx - n_y - 4]; // W, E, S, N

     immNeigh = 0.25*(uc[idx - n_y - 5] + uc[idx - n_y - 3] + uc[idx + n_y + 3] + uc[idx + n_y + 5]);  // NW, NE, SW, SE;

     NeighNeigh = 0.125*(uc[idx-2] + uc[idx+2] + uc[idx - 2*(n_y + 4)] - uc[idx + 2*(n_y + 4)]);  // WW, EE, NN, SS;

     un[idx] = 2*uc[idx] - uo[idx] + VSQR *(dt * dt) * (Neigh + immNeigh + NeighNeigh - 5.5 * uc[idx])/(h * h) + f_pebble(pebbles[idx_p_1],t);
   }

}

void run_gpu(double *u, double *u0, double *u1, double *pebbles, double h, double end_time, int nthreads)
{
	hipEvent_t kstart, kstop;
	float ktime;
  int BLKS_X, BLKS_Y;

	/* HW2: Define your local variables here CPU Side */

  double t;
  double dt;
  t = 0.;
  dt = h / 2.;

  double *uc, *uo ; // Host Side mem pointers

  //un = (double*)calloc(narea, sizeof(double));
  uc = (double*)calloc(narea, sizeof(double));
  uo = (double*)calloc(narea, sizeof(double));
	//pb = (double*)calloc(npoints_y*npoints_x, sizeof(double));

  /* Set up device timers */
  CUDA_CALL(hipSetDevice(0));
  CUDA_CALL(hipEventCreate(&kstart));
  CUDA_CALL(hipEventCreate(&kstop));

  double *un_cuda,*uc_cuda, *uo_cuda, *pb; // Device Side Memory Access pointers


  /* HW2: Add CUDA kernel call preperation code here */

  BLKS_Y = npoints_y/nthreads;
  BLKS_Y += npoints_y%nthreads ? 1 : 0;

  BLKS_X = npoints_x/nthreads;
  BLKS_X += npoints_x%nthreads ? 1 : 0;

  hipMalloc((void**)&un_cuda, (narea)*sizeof(double));
  hipMalloc((void**)&uc_cuda, (narea)*sizeof(double));
  hipMalloc((void**)&uo_cuda, (narea)*sizeof(double));
  hipMalloc((void**)&pb, (npoints_x*npoints_y)*sizeof(double));

  hipMemcpy(uc_cuda, u1, sizeof(double)*narea, hipMemcpyHostToDevice);
  hipMemcpy(uo_cuda, u0, sizeof(double)*narea, hipMemcpyHostToDevice);
  hipMemcpy(pb, pebbles, sizeof(double)*(npoints_x*npoints_y), hipMemcpyHostToDevice);


  dim3 block_dim(nthreads, nthreads);
  dim3 grid_dim(BLKS_X, BLKS_Y);


	/* Start GPU computation timer */
	CUDA_CALL(hipEventRecord(kstart, 0));

	/* HW2: Add main lake simulation loop here */
  while(1)
  {
    evolve13pt_gpu<<<grid_dim,block_dim>>>(un_cuda, uc_cuda, uo_cuda, pb, npoints_x, npoints_y, h, dt, t);

       hipMemcpy(uc, un_cuda, sizeof(double)*narea, hipMemcpyDeviceToHost);
  	   hipMemcpy(uo, uc_cuda, sizeof(double)*narea, hipMemcpyDeviceToHost);

       hipMemcpy(uc_cuda, uc,  sizeof(double)*narea, hipMemcpyHostToDevice);
       hipMemcpy(uo_cuda, uo,  sizeof(double)*narea, hipMemcpyHostToDevice);

      if(!tpdt(&t,dt,end_time)) break;
  }

  hipMemcpy(u, un_cuda, sizeof(double), hipMemcpyDeviceToHost);

     /* Stop GPU computation timer */
	CUDA_CALL(hipEventRecord(kstop, 0));
	CUDA_CALL(hipEventSynchronize(kstop));
	CUDA_CALL(hipEventElapsedTime(&ktime, kstart, kstop));
	printf("GPU computation: %f msec\n", ktime);

	/* HW2: Add post CUDA kernel call processing and cleanup here */
  hipFree(un_cuda);
  hipFree(uc_cuda);
  hipFree(uo_cuda);
  hipFree(pb);

	/* timer cleanup */
	CUDA_CALL(hipEventDestroy(kstart));
	CUDA_CALL(hipEventDestroy(kstop));

  free(uc);
  free(uo);
  //free(pb);
}
